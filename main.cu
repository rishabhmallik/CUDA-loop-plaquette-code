#include "hip/hip_runtime.h"
//cuda loop-plaquette code
#include <iostream>
#include <math.h>
#include <random>
#include <time.h>
#include <chrono>
#include <fstream>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include "parameters.h"

using namespace std::chrono;
using namespace std;

__host__ __device__ int mod(int a,int b){
    return (a%b + b)%b;
}

__device__ void shiftx(int b[], int a[], int sign, int v){
    for (int i=0; i<d; i++){
        if (i==v){
            if (v==0){ b[i]=mod(a[i]+sign,Nt);  }
            else { b[i]=mod(a[i]+sign,Ns);  }
        }
        else { b[i]=a[i]; }
    }
}

double I(int s){
    double a=0,r=0;
    while(r<inf){
        a+=dr*pow(r,s+1)*exp(-eta*pow(r,2)-lmd*pow(r,4));
        r+=dr;
    }
    return a;
}

__host__ __device__ int sx(int x[], int k[], int a[]){
    int sum=0;
    int v[d]={0};
    for (int i=0;i<d;i++){
        v[i]=1;
        
        sum+=abs(k[x[0]+Nt*x[1]+Nt*Ns*i])
        +abs(k[mod(x[0]-v[0],Nt)+Nt*mod(x[1]-v[1],Ns)+Nt*Ns*i])
        +2*(a[x[0]+Nt*x[1]+Nt*Ns*i]
        +a[mod(x[0]-v[0],Nt)+Nt*mod(x[1]-v[1],Ns)+Nt*Ns*i]);

        v[i]=0;
    }
    return sum;
}
__device__ int countl(int *l){
    int sum=0;
    for (int i=0; i<Nt; i++){
        for (int j=0; j<Ns; j++){
            for (int m=0; m<d; m++) sum+=abs(l[i+j*Nt+m*Nt*Ns]);
        }
    }
    return sum;
}
__global__ void a_update(int t, int tag, int *k, int *a, int *a_, double *I_val){
    int y, x[d], x_[d];
    double rho;
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    
    if (id>Ns*Nt/2-1) return;
    if (t==0){
        x[0]=(2*id+tag)%Nt;
        x[1]=(2*id+tag)/Nt;
    }
    else {
        x[0]=(2*id+tag)/Ns;
        x[1]=(2*id+tag)%Ns;
    }
    
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    double rand1 = hiprand_uniform_double(&state);
    int rand2 = 2*((int)(2*(1-hiprand_uniform_double(&state))))-1;
    
    y=a_[x[0]+Nt*x[1]+Nt*Ns*t];
    a_[x[0]+Nt*x[1]+Nt*Ns*t]=a[x[0]+Nt*x[1]+Nt*Ns*t]+rand2;
    
    if (a_[x[0]+Nt*x[1]+Nt*Ns*t]<0){
        a_[x[0]+Nt*x[1]+Nt*Ns*t]=y;
        return;
    }
    
    shiftx(x_, x, 1, t);
    if (a_[x[0]+Nt*x[1]+Nt*Ns*t]>a[x[0]+Nt*x[1]+Nt*Ns*t]){
        rho=1.0/(abs(k[x[0]+Nt*x[1]+Nt*Ns*t])+a_[x[0]+Nt*x[1]+Nt*Ns*t])
        /a_[x[0]+Nt*x[1]+Nt*Ns*t]
        *I_val[sx(x,k,a_)]*I_val[sx(x_,k,a_)]
        /I_val[sx(x,k,a)]/I_val[sx(x_,k,a)];
    } 
    else{
        rho=1.0*(abs(k[x[0]+Nt*x[1]+Nt*Ns*t])+a[x[0]+Nt*x[1]+Nt*Ns*t])
        *a[x[0]+Nt*x[1]+Nt*Ns*t]
        *I_val[sx(x,k,a_)]*I_val[sx(x_,k,a_)]
        /I_val[sx(x,k,a)]/I_val[sx(x_,k,a)];
    }
    
    if (rand1<rho){
        a[x[0]+Nt*x[1]+Nt*Ns*t]=a_[x[0]+Nt*x[1]+Nt*Ns*t];
    }
    else{
        a_[x[0]+Nt*x[1]+Nt*Ns*t]=y;
    }
    
}
__global__ void plaquette_update(int i, int j, int *k, int *a, double *I_val){
    int x[d], x1[d], x2[d], x12[d], del;
    double rho=1.0;
    
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    if (id>Ns*Nt/4-1) return;
    x[0]=(2*id+i)%Nt;
    x[1]=((2*id+i)/Nt)*2+j;
    
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    double rand1 = hiprand_uniform_double(&state);
    int rand2 = 2*((int)(2*(1-hiprand_uniform_double(&state))))-1;
    
    del=rand2;
    
    shiftx(x2,x,1,1);
    shiftx(x1,x,1,0);
    shiftx(x12,x1,1,1);
    
    if (del>0)
    {
        if (k[x[0]+Nt*x[1]+Nt*Ns]>=0) rho*=1.0/(k[x[0]+Nt*x[1]+Nt*Ns]+1+a[x[0]+Nt*x[1]+Nt*Ns]);
        else rho*=abs(k[x[0]+Nt*x[1]+Nt*Ns])+a[x[0]+Nt*x[1]+Nt*Ns];
        if (k[x2[0]+Nt*x2[1]]>=0) rho*=1.0/(k[x2[0]+Nt*x2[1]]+1+a[x2[0]+Nt*x2[1]]);
        else rho*=abs(k[x2[0]+Nt*x2[1]])+a[x2[0]+Nt*x2[1]];
        if (k[x[0]+Nt*x[1]]>0) rho*=k[x[0]+Nt*x[1]]+a[x[0]+Nt*x[1]];
        else rho*=1.0/(abs(k[x[0]+Nt*x[1]])+1+a[x[0]+Nt*x[1]]);
        if (k[x1[0]+Nt*x1[1]+Nt*Ns]>0) rho*=k[x1[0]+Nt*x1[1]+Nt*Ns]+a[x1[0]+Nt*x1[1]+Nt*Ns];
        else rho*=1.0/(abs(k[x1[0]+Nt*x1[1]+Nt*Ns])+1+a[x1[0]+Nt*x1[1]+Nt*Ns]);
    
        if (k[x[0]+Nt*x[1]+Nt*Ns]>=0 && k[x[0]+Nt*x[1]]<=0) rho*=I_val[sx(x,k,a)+2]/I_val[sx(x,k,a)];
        else if (k[x[0]+Nt*x[1]+Nt*Ns]<0 && k[x[0]+Nt*x[1]]>0) rho*=I_val[sx(x,k,a)-2]/I_val[sx(x,k,a)];
        if (k[x2[0]+Nt*x2[1]]>=0 && k[x1[0]+Nt*x1[1]+Nt*Ns]<=0) rho*=I_val[sx(x12,k,a)+2]/I_val[sx(x12,k,a)];
        else if (k[x2[0]+Nt*x2[1]]<0 && k[x1[0]+Nt*x1[1]+Nt*Ns]>0) rho*=I_val[sx(x12,k,a)-2]/I_val[sx(x12,k,a)];
        if (k[x2[0]+Nt*x2[1]]>=0 && k[x[0]+Nt*x[1]+Nt*Ns]>=0) rho*=I_val[sx(x2,k,a)+2]/I_val[sx(x2,k,a)];
        else if (k[x2[0]+Nt*x2[1]]<0 && k[x[0]+Nt*x[1]+Nt*Ns]<0) rho*=I_val[sx(x2,k,a)-2]/I_val[sx(x2,k,a)];
        if (k[x1[0]+Nt*x1[1]+Nt*Ns]<=0 && k[x[0]+Nt*x[1]]<=0) rho*=I_val[sx(x1,k,a)+2]/I_val[sx(x1,k,a)];
        else if(k[x1[0]+Nt*x1[1]+Nt*Ns]>0 && k[x[0]+Nt*x[1]]>0) rho*=I_val[sx(x1,k,a)-2]/I_val[sx(x1,k,a)];
    } 
    else
    {
        if (k[x[0]+Nt*x[1]+Nt*Ns]<=0) rho*=1.0/(abs(k[x[0]+Nt*x[1]+Nt*Ns])+1+a[x[0]+Nt*x[1]+Nt*Ns]);
        else rho*=k[x[0]+Nt*x[1]+Nt*Ns]+a[x[0]+Nt*x[1]+Nt*Ns];
        if (k[x2[0]+Nt*x2[1]]<=0) rho*=1.0/(abs(k[x2[0]+Nt*x2[1]])+1+a[x2[0]+Nt*x2[1]]);
        else rho*=k[x2[0]+Nt*x2[1]]+a[x2[0]+Nt*x2[1]];
        if (k[x[0]+Nt*x[1]]<0)rho*=abs(k[x[0]+Nt*x[1]])+a[x[0]+Nt*x[1]];
        else rho*=1.0/(k[x[0]+Nt*x[1]]+1+a[x[0]+Nt*x[1]]);
        if (k[x1[0]+Nt*x1[1]+Nt*Ns]<0) rho*=abs(k[x1[0]+Nt*x1[1]+Nt*Ns])+a[x1[0]+Nt*x1[1]+Nt*Ns];
        else rho*=1.0/(k[x1[0]+Nt*x1[1]+Nt*Ns]+1+a[x1[0]+Nt*x1[1]+Nt*Ns]);
    
        if (k[x[0]+Nt*x[1]+Nt*Ns]<=0 && k[x[0]+Nt*x[1]]>=0) rho*=I_val[sx(x,k,a)+2]/I_val[sx(x,k,a)];
        else if (k[x[0]+Nt*x[1]+Nt*Ns]>0 && k[x[0]+Nt*x[1]]<0) rho*=I_val[sx(x,k,a)-2]/I_val[sx(x,k,a)];
        if (k[x2[0]+Nt*x2[1]]<=0 && k[x1[0]+Nt*x1[1]+Nt*Ns]>=0) rho*=I_val[sx(x12,k,a)+2]/I_val[sx(x12,k,a)];
        else if (k[x2[0]+Nt*x2[1]]>0 && k[x1[0]+Nt*x1[1]+Nt*Ns]<0) rho*=I_val[sx(x12,k,a)-2]/I_val[sx(x12,k,a)];
        if (k[x2[0]+Nt*x2[1]]<=0 && k[x[0]+Nt*x[1]+Nt*Ns]<=0) rho*=I_val[sx(x2,k,a)+2]/I_val[sx(x2,k,a)];
        else if (k[x2[0]+Nt*x2[1]]>0 && k[x[0]+Nt*x[1]+Nt*Ns]>0) rho*=I_val[sx(x2,k,a)-2]/I_val[sx(x2,k,a)];
        if (k[x1[0]+Nt*x1[1]+Nt*Ns]>=0 && k[x[0]+Nt*x[1]]>=0) rho*=I_val[sx(x1,k,a)+2]/I_val[sx(x1,k,a)];
        else if(k[x1[0]+Nt*x1[1]+Nt*Ns]<0 && k[x[0]+Nt*x[1]]<0) rho*=I_val[sx(x1,k,a)-2]/I_val[sx(x1,k,a)];
    }
    if (rand1<rho)
    {
        k[x[0]+Nt*x[1]+Nt*Ns]+=del;
        k[x2[0]+Nt*x2[1]]+=del;
        k[x1[0]+Nt*x1[1]+Nt*Ns]-=del;
        k[x[0]+Nt*x[1]]-=del;
    }
}
__global__ void delrandvals(int *delrand, int N){
    
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    if (id>N-1) return;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    delrand[id] = 2*((int)(2*(1-hiprand_uniform_double(&state))))-1;
}
__global__ void k_rho(int t, int *delrand, double *rho, int *k, int *a, double mu, double *I_val) {
    
    int x[d], x_[d];
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    if (id>Ns*Nt-1) return;
    x[0]=id%Nt;
    x[1]=id/Nt;
    rho[x[0]+Nt*x[1]]=1.0;
    int del=delrand[x[1-t]];
    shiftx(x_,x,-1,t);
    
    if (del>0)
    {
        if (k[x[0]+Nt*x[1]+Nt*Ns*t]>=0) rho[x[0]+Nt*x[1]]*=1.0/(k[x[0]+Nt*x[1]+Nt*Ns*t]+1+a[x[0]+Nt*x[1]+Nt*Ns*t]);
        else rho[x[0]+Nt*x[1]]*=abs(k[x[0]+Nt*x[1]+Nt*Ns*t])+a[x[0]+Nt*x[1]+Nt*Ns*t];
        if (k[x[0]+Nt*x[1]+Nt*Ns*t]>=0 && k[x_[0]+Nt*x_[1]+Nt*Ns*t]>=0) rho[x[0]+Nt*x[1]]*=I_val[sx(x,k,a)+2]/I_val[sx(x,k,a)];
        else if (k[x[0]+Nt*x[1]+Nt*Ns*t]<0 && k[x_[0]+Nt*x_[1]+Nt*Ns*t]<0) rho[x[0]+Nt*x[1]]*=I_val[sx(x,k,a)-2]/I_val[sx(x,k,a)];
    }
    else
    {
        if (k[x[0]+Nt*x[1]+Nt*Ns*t]>0) rho[x[0]+Nt*x[1]]*=k[x[0]+Nt*x[1]+Nt*Ns*t]+a[x[0]+Nt*x[1]+Nt*Ns*t];
        else rho[x[0]+Nt*x[1]]*=1.0/(abs(k[x[0]+Nt*x[1]+Nt*Ns*t])+a[x[0]+Nt*x[1]+Nt*Ns*t]+1);
        if (k[x[0]+Nt*x[1]+Nt*Ns*t]<=0 && k[x_[0]+Nt*x_[1]+Nt*Ns*t]<=0) rho[x[0]+Nt*x[1]]*=I_val[sx(x,k,a)+2]/I_val[sx(x,k,a)];
        else if (k[x[0]+Nt*x[1]+Nt*Ns*t]>0 && k[x_[0]+Nt*x_[1]+Nt*Ns*t]>0) rho[x[0]+Nt*x[1]]*=I_val[sx(x,k,a)-2]/I_val[sx(x,k,a)];
    }
    rho[id]*=exp(del*mu*(t==0));
}
__global__ void temporalloop_update(double *rho_vals, int *rand2, int *k){
    
    double rho=1.0;
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    if (id>Ns-1) return;
    
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    double rand1 = hiprand_uniform_double(&state);
    
    for (int bx=0; bx<Nt; bx++){
        rho*=rho_vals[bx+Nt*id];
    }
    if (rand1<rho){
        for (int bx=0; bx<Nt; bx++){
            k[bx+Nt*id]+=rand2[id];
        }
    }
    
}
__global__ void spatialloop_update(double *rho_vals, int *rand2, int *k){
    
    double rho=1.0;
    int id=threadIdx.x + blockDim.x * blockIdx.x;
    if (id>Nt-1) return;
    
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    double rand1 = hiprand_uniform_double(&state);
    
    for (int by=0; by<Ns; by++){
        rho*=rho_vals[id+Nt*by];
    }
    if (rand1<rho){
        for (int by=0; by<Ns; by++){
            k[id+Nt*by+Nt*Ns]+=rand2[id];
        }
    }
}

void update(int *k, int *a, int *a_, double mu, double *I_val, bool flag2){
    
    blocks=Ns*Nt/2/blockSize+1;
    
    a_update<<<blocks,blockSize>>>(0,0,k,a,a_,I_val);
    a_update<<<blocks,blockSize>>>(0,1,k,a,a_,I_val);
    a_update<<<blocks,blockSize>>>(1,0,k,a,a_,I_val);
    a_update<<<blocks,blockSize>>>(1,1,k,a,a_,I_val);
    
    blocks=Ns*Nt/4/blockSize+1;
    plaquette_update<<<blocks,blockSize>>>(0,0,k,a,I_val);
    plaquette_update<<<blocks,blockSize>>>(0,1,k,a,I_val);
    plaquette_update<<<blocks,blockSize>>>(1,0,k,a,I_val);
    plaquette_update<<<blocks,blockSize>>>(1,1,k,a,I_val);
    
    int *delrand;
    double *rho;

    hipMalloc(&rho, Nt*Ns*sizeof(*rho));
    hipMalloc(&delrand, Ns*sizeof(*delrand));
	
    blocks=Ns/blockSize+1;
    delrandvals<<<blocks,blockSize>>>(delrand,Ns);
    
    blocks=Ns*Nt/blockSize+1;
    k_rho<<<blocks,blockSize>>>(0,delrand,rho,k,a,mu,I_val);
    
    blocks=Ns/blockSize+1;
    temporalloop_update<<<blocks,blockSize>>>(rho,delrand,k);
    hipFree(delrand);
    
    hipMalloc(&delrand, Nt*sizeof(*delrand));
    
    blocks=Nt/blockSize+1;
    delrandvals<<<blocks,blockSize>>>(delrand,Nt);
    
    blocks=Ns*Nt/blockSize+1;
    k_rho<<<blocks,blockSize>>>(1,delrand,rho,k,a,mu,I_val);
    
    blocks=Nt/blockSize+1;
    spatialloop_update<<<blocks,blockSize>>>(rho,delrand,k);
    
}

__global__ void init_lattice(int *A, int a, int n){
    int i=blockDim.x*blockIdx.x+threadIdx.x;
    if (i<n) A[i]=a;
}


__global__ void arraycopy(int *A, int *B, int n){
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    if (id<n) A[id]=B[id];
}
int ksum(int k[]){
    int sum=0;
    for (int i=0;i<Nt;i++){
        for (int j=0;j<Ns;j++){
            sum+=k[i+j*Nt];
        }
    }
    return sum;
}

double phi2(int k[], int a[], double I[int_val]){
    double sum=0;
    int x[d];
    for (int i=0;i<Nt;i++){
        for (int j=0;j<Ns;j++){
            x[0]=i; x[1]=j;
            sum+=I[sx(x, k, a)+2]/I[sx(x, k, a)];
        }
    }
    return sum/Nt/Ns;
}
double phi4(int k[], int a[], double I[int_val]){
    double sum=0;
    int x[d];
    for (int i=0;i<Nt;i++){
        for (int j=0;j<Ns;j++){
            x[0]=i; x[1]=j;
            sum+=I[sx(x, k, a)+4]/I[sx(x, k, a)];
        }
    
    }
    return sum/Nt/Ns;
}

double errorjack(double *xi, int configs){
    double *x_i, x_=0, stddev=0;
    x_i=(double*) malloc(configs*sizeof(*x_i));
    for (int i=0; i<configs; i++){
        x_i[i]=0;
        for (int j=0; j<configs; j++){
            x_i[i]+=(1-(i==j))*xi[j];
        }
        x_i[i]=x_i[i]/(configs-1);
        x_+=x_i[i];
    }
    x_=x_/configs;
    for (int i=0; i<configs; i++){
        stddev+=(x_i[i]-x_)*(x_i[i]-x_);
    }
    stddev=sqrt(stddev*(configs-1)/configs);
    return stddev;
}

int main(int argc, char **argv)
{
    int *k, *a, *a_, *kh, *ah, *a_h;
    double *I_val;
    
    hipMalloc(&k, Nt*Ns*d*sizeof(*k));
    hipMalloc(&a, Nt*Ns*d*sizeof(*a));
    hipMalloc(&a_, Nt*Ns*d*sizeof(*a_));
    hipMalloc(&I_val, int_val*sizeof(*I_val));
    
    kh=(int*) malloc(Nt*Ns*d*sizeof(*kh));
    ah=(int*) malloc(Nt*Ns*d*sizeof(*ah));
    a_h=(int*) malloc(Nt*Ns*d*sizeof(*a_h));
    
    double dmu=(mu_max-mu_min)/mu_n; 
    double n_avg, phi2_avg, phi4_avg;
    double *xi,*phi2i, I_val_h[int_val];
    double mu=mu_max-dmu;
    
    xi=(double*) malloc(configs*sizeof(*xi));
    phi2i=(double*) malloc(configs*sizeof(*phi2i));
    
    for (int i=0; i<int_val; i++){
        I_val_h[i]=I(i);
    }
    hipMemcpy(I_val, I_val_h, int_val*sizeof(*I_val), hipMemcpyHostToDevice);
    ofstream data, data1, data2, data4, data5;
    
	string filename1="mu_vs_n.txt"
    ,filename2="mu_vs_phi2.txt";
	
    data1.open(filename1);
    data2.open(filename2);
    
    blocks=Nt*Ns*d/blockSize+1;
    
    auto begin=high_resolution_clock::now();
    
    for (int g=0; g<mu_n; g++){
	
		init_lattice<<<blocks,blockSize>>>(k,0,Nt*Ns*d);
		init_lattice<<<blocks,blockSize>>>(a,0,Nt*Ns*d);
		init_lattice<<<blocks,blockSize>>>(a_,0,Nt*Ns*d);
        double mu_therm=1.3;
        for (int i=0; i<equil; i++){
            update(k,a,a_,mu_therm,I_val,false); 
        }
		
		for (int i=0; i<equil; i++){
            update(k,a,a_,mu,I_val,false); 
        }
        phi2_avg=0;
        n_avg=0;
        
        for (int i=0; i<configs; i++){
            for (int j=0; j<gaps; j++){
                update(k,a,a_,mu,I_val,true);
            }
            update(k,a,a_,mu,I_val,true);
            hipDeviceSynchronize();
            
            hipMemcpy(kh, k, Nt*Ns*d*sizeof(*k), hipMemcpyDeviceToHost);
            hipMemcpy(ah, a, Nt*Ns*d*sizeof(*a), hipMemcpyDeviceToHost);
            
			xi[i]=1.0*ksum(kh)/Nt/Ns;
            phi2i[i]=phi2(kh,ah,I_val_h);
            
            n_avg+=xi[i];
            phi2_avg+=phi2i[i];
        }
        n_avg=n_avg/configs;
        phi2_avg=phi2_avg/configs;
        
        data1<<mu<<"\t"<<n_avg<<"\t"<<errorjack(xi,configs)<<"\n";
        data2<<mu<<"\t"<<phi2_avg<<"\t"<<errorjack(phi2i,configs)<<"\n";
        mu-=dmu;
        cout<<g<<endl;
	
    } 
    
    data1.close();
    data2.close();
    hipFree(k);
    hipFree(a);
    hipFree(a_);
    hipFree(I_val);
    free(kh);
    free(ah);
    free(a_h);
    
    auto stop=high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop-begin);
    
    cout<<duration.count()<<endl;
    
    return 0;
}
